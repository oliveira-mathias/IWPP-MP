#include "hip/hip_runtime.h"
#include "MorphologicalReconstruction.cuh"

unsigned int __device__ atomicCASUChar(unsigned char* address, unsigned int index, unsigned char assumed, unsigned char val) {
    unsigned int assumed_word;
    unsigned int val_word;
    unsigned char* val_word_as_uchar;
    unsigned char* assumed_word_as_uchar;
    unsigned int* address_as_uint;

    unsigned int word_index, byte_offset;

    // Endereçamento
    word_index = index/sizeof(unsigned int);
    byte_offset = index - word_index*sizeof(unsigned int);

    address_as_uint = (unsigned int*)(&address[word_index*sizeof(unsigned int)]);
    assumed_word_as_uchar = (unsigned char*) &assumed_word;
    val_word_as_uchar = (unsigned char*) &val_word;
    
    // Leitura das palavras da memória
    assumed_word = *address_as_uint;
    val_word = assumed_word;

    // Atualizando os valores
    assumed_word_as_uchar[byte_offset] = assumed;
    val_word_as_uchar[byte_offset] = val;

    val_word = atomicCAS(address_as_uint, assumed_word, val_word);

    return val_word;
}


bool __device__ XForwardPropagation(unsigned char mp_marker[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2], 
                                unsigned char mp_mask[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2]) 
{
    int tid = threadIdx.x;
    int elementsUpdated = 0;
    int j;
    int myBonusRow;

    for(j=0; j<MEGAPIXEL_SIZE+1; j++){
        if((mp_marker[tid+1][j] > mp_marker[tid+1][j+1]) and (mp_marker[tid+1][j+1] < mp_mask[tid+1][j+1])) {
            mp_marker[tid+1][j+1] = min(mp_marker[tid+1][j], mp_mask[tid+1][j+1]);
            elementsUpdated = 1;
        }
    }

    // Laterais superior e inferior
    myBonusRow = (MEGAPIXEL_SIZE+1)*tid;
    if(tid==0 or tid==1) {
        for(j=0; j<MEGAPIXEL_SIZE+1; j++){
            if(mp_marker[myBonusRow][j] > mp_marker[myBonusRow][j+1] and mp_marker[myBonusRow][j+1] < mp_mask[myBonusRow][j+1]) {
                mp_marker[myBonusRow][j+1] = min(mp_marker[myBonusRow][j], mp_mask[myBonusRow][j+1]);
                elementsUpdated = 1;
            }
        }
    }

    // Verificamos se algum elemento foi atualizado
    elementsUpdated = __syncthreads_or(elementsUpdated);

    return (elementsUpdated != 0);
}

bool __device__ XBackwardPropagation(unsigned char mp_marker[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2], 
                                unsigned char mp_mask[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2]) 
{
    int tid = threadIdx.x;
    int elementsUpdated = 0;
    int j;
    int myBonusRow;

    for(j=MEGAPIXEL_SIZE+1; j>0; j--){
        if(mp_marker[tid+1][j] > mp_marker[tid+1][j-1] and mp_marker[tid+1][j-1] < mp_mask[tid+1][j-1]) {
            mp_marker[tid+1][j-1] = min(mp_marker[tid+1][j], mp_mask[tid+1][j-1]);
            elementsUpdated = 1;
        }
    }

    // Laterais superior e inferior
    myBonusRow = (MEGAPIXEL_SIZE+1)*tid;
    if(tid==0 or tid==1) {
        for(j=MEGAPIXEL_SIZE+1; j>0; j--){
            if(mp_marker[myBonusRow][j] > mp_marker[myBonusRow][j-1] and mp_marker[myBonusRow][j-1] < mp_mask[myBonusRow][j-1]) {
                mp_marker[myBonusRow][j-1] = min(mp_marker[myBonusRow][j], mp_mask[myBonusRow][j-1]);
                elementsUpdated = 1;
            }
        }
    }

    // Verificamos se algum elemento foi atualizado
    elementsUpdated = __syncthreads_or(elementsUpdated);

    return (elementsUpdated != 0);
}

bool __device__ YUpwardPropagation(unsigned char mp_marker[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2], 
                                unsigned char mp_mask[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2])
{
    int tid = threadIdx.x;
    int elementsUpdated = 0;
    int i;
    int myBonusCol;

    for(i=MEGAPIXEL_SIZE+1; i>0; i--){
        if(mp_marker[i][tid+1] > mp_marker[i-1][tid+1] and mp_marker[i-1][tid+1] < mp_mask[i-1][tid+1]) {
            mp_marker[i-1][tid+1] = min(mp_marker[i][tid+1], mp_mask[i-1][tid+1]);
            elementsUpdated = 1;
        }
    }

    // Laterais superior e inferior
    myBonusCol = (MEGAPIXEL_SIZE+1)*tid;
    if(tid==0 or tid==1) {
        for(i=MEGAPIXEL_SIZE+1; i>0; i--){
            if(mp_marker[i][myBonusCol] > mp_marker[i-1][myBonusCol] and mp_marker[i-1][myBonusCol] < mp_mask[i-1][myBonusCol]) {
                mp_marker[i-1][myBonusCol] = min(mp_marker[i][myBonusCol], mp_mask[i-1][myBonusCol]);
                elementsUpdated = 1;
            }
        }
    }

    // Verificamos se algum elemento foi atualizado
    elementsUpdated = __syncthreads_or(elementsUpdated);

    return (elementsUpdated != 0);   
}

// Note que a segunda condição do if pode ser removida
bool __device__ YDownwardPropagation(unsigned char mp_marker[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2], 
                                unsigned char mp_mask[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2])
{
    int tid = threadIdx.x;
    int elementsUpdated = 0;
    int i;
    int myBonusCol;

    for(i=0; i<MEGAPIXEL_SIZE+1; i++){
        if(mp_marker[i][tid+1] > mp_marker[i+1][tid+1] and mp_marker[i+1][tid+1] < mp_mask[i+1][tid+1]) {
            mp_marker[i+1][tid+1] = min(mp_marker[i][tid+1], mp_mask[i+1][tid+1]);
            elementsUpdated = 1;
        }
    }

    // Laterais superior e inferior
    myBonusCol = (MEGAPIXEL_SIZE+1)*tid;
    if(tid==0 or tid==1) {
        for(i=0; i<MEGAPIXEL_SIZE+1; i++){
            if(mp_marker[i][myBonusCol] > mp_marker[i+1][myBonusCol] and mp_marker[i+1][myBonusCol] < mp_mask[i+1][myBonusCol]) {
                mp_marker[i+1][myBonusCol] = min(mp_marker[i][myBonusCol], mp_mask[i+1][myBonusCol]);
                elementsUpdated = 1;
            }
        }
    }

    // Verificamos se algum elemento foi atualizado
    elementsUpdated = __syncthreads_or(elementsUpdated);

    return (elementsUpdated != 0);
}

// Pode apresentar overflow na fila de megapixels se a imagem possuir pelo menos 65536 megapixels de altura/largura
void __global__ MorphologicalReconstructionKernel(GlobalQueue* gq, volatile unsigned char* marker, unsigned char* mask, 
            size_t pitchMarker, size_t pitchMask, int size, int* memLeak)
{
    // Assume que o bloco tem a forma (1,32)
    int tid = threadIdx.x;
    int blockId = blockIdx.x;
    int numBlocks = gridDim.x;

    // Variáveis do MegaPixel
    volatile unsigned char* markerRow;
    unsigned char* maskRow;
    __shared__ unsigned char mp_marker[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2];
    __shared__ unsigned char mp_mask[MEGAPIXEL_SIZE+2][MEGAPIXEL_SIZE+2];

    // Variáveis fila global
    // 1KB de buffer de leitura e de escrita
    __shared__ ushort2 readBuffer[BUFFER_SIZE];
    __shared__ ushort2 writeBuffer[BUFFER_SIZE];

    // Variáveis auxiliares de leitura e escrita
    __shared__ int writeSize;
    __shared__ int readOffset;
    int readedAmount;
    int writtenAmount;
    int currentRegister;

    // Fila de leitura
    ushort2* readGlobalQueue;
    int readQueueSize;

     // Variáveis auxiliares
    int i, j;

    // Variáveis de propagação
    int elementsUpdated;


    // Write back to memory variables
    int rowOffset = tid/8; 
    int wordOffset = (tid%8)*4;
    unsigned int val, assumed;
    unsigned char* valAsChar = (unsigned char*) &val;

    int countElementsUpdated;

    // Variáveis de coleta de tempo
    // unsigned long long queueRead=0;
    // unsigned long long MP_Fetch=0;
    // unsigned long long MP_Processing=0;
    // unsigned long long MP_Dump=0;
    // unsigned long long queueWrite=0;

    // unsigned long long auxStart;


    // Incializando a fila do bloco e primitiva de sincronização
    if(tid==0) {
        writeSize = 0;
        readOffset = 0;
    }

    // Inicializando a fila de leitura
    const int quotient = (gq->readQueueSize)/numBlocks;
    const int remainder = (gq->readQueueSize)%numBlocks;
    
    // Tamanho da fila
    readQueueSize = quotient;
    if(blockId < remainder) {
        readQueueSize++;
    }

    // Offset
    int offset = min(remainder, blockId)*(quotient + 1);
    if(blockId > remainder) {
        offset += (blockId - remainder)*quotient;
    }

    // Ponteiro para a fila
    readGlobalQueue = &(gq->readQueue[offset]);

    // Sincronia para garantia de consistência de memória compartilhada
    __syncthreads();

    
    // Processamento da fila global
    // auxStart = clock64();
    while(readedAmount = readFromGlobalQueue(readGlobalQueue, readOffset, readQueueSize, readBuffer, BUFFER_SIZE)) {
      // queueRead += clock64() - auxStart;

        // Processamos os registros lidos da fila global
        for(currentRegister=0; currentRegister<readedAmount; currentRegister++) {


            // Realizando o fetch da memória global para o megapixel
            // Leitura do bloco principal
            // auxStart = clock64();
            #pragma unroll
            for(i=0; i<MEGAPIXEL_SIZE; i++) {
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + i) * pitchMarker);
                maskRow = (unsigned char*)((char*)mask + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + i) * pitchMask);

                mp_marker[i+1][tid+1] = markerRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];
                mp_mask[i+1][tid+1] = maskRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];

            }

            // Leitura da lateral superior
            if(readBuffer[currentRegister].x > 0) {
                // Ponteiro para o megapixel acima
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x - 1) * pitchMarker);
                maskRow = (unsigned char*)((char*)mask + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x - 1) * pitchMask);

                mp_marker[0][tid+1] = markerRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];
                mp_mask[0][tid+1] = maskRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];
            }
            else {
                // Acho que (0,0) funciona para marcar um pixel que não deve interagir com os demais
                mp_marker[0][tid+1] = 0;
                mp_mask[0][tid+1] = 0;
            }

            // Leitura da lateral inferior
            if(readBuffer[currentRegister].x < (size/MEGAPIXEL_SIZE)-1) {
                // Ponteiro para o megapixel abaixo
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*(readBuffer[currentRegister].x + 1)) * pitchMarker);
                maskRow = (unsigned char*)((char*)mask + (MEGAPIXEL_SIZE*(readBuffer[currentRegister].x + 1)) * pitchMask);

                mp_marker[MEGAPIXEL_SIZE+1][tid+1] = markerRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];
                mp_mask[MEGAPIXEL_SIZE+1][tid+1] = maskRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y + tid];
            }
            else {
                mp_marker[MEGAPIXEL_SIZE+1][tid+1] = 0;
                mp_mask[MEGAPIXEL_SIZE+1][tid+1] = 0;
            }

            // Leitura da lateral direita
            if(readBuffer[currentRegister].y < (size/MEGAPIXEL_SIZE)-1) {
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMarker);
                maskRow = (unsigned char*)((char*)mask + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMask);

                mp_marker[tid+1][MEGAPIXEL_SIZE+1] = markerRow[MEGAPIXEL_SIZE*(readBuffer[currentRegister].y+1)];
                mp_mask[tid+1][MEGAPIXEL_SIZE+1] = maskRow[MEGAPIXEL_SIZE*(readBuffer[currentRegister].y+1)];
            }
            else {
                mp_marker[tid+1][MEGAPIXEL_SIZE+1] = 0;
                mp_mask[tid+1][MEGAPIXEL_SIZE+1] = 0;
            }

            // Leitura da lateral esquerda
            if(readBuffer[currentRegister].y > 0) {
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMarker);
                maskRow = (unsigned char*)((char*)mask + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMask);

                mp_marker[tid+1][0] = markerRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y - 1];
                mp_mask[tid+1][0] = maskRow[MEGAPIXEL_SIZE*readBuffer[currentRegister].y - 1];
            }
            else {
                mp_marker[tid+1][0] = 0;
                mp_mask[tid+1][0] = 0;
            }
            // MP_Fetch += clock64() - auxStart;

            // Inicializamos as quinas pra evitar problemas durante a propagação
            // Pode ser otimizado para explorar o paralelismo
            if(tid==0) {
                mp_marker[0][0] = 0;
                mp_marker[0][MEGAPIXEL_SIZE+1] = 0;
                mp_marker[MEGAPIXEL_SIZE+1][0] = 0;
                mp_marker[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE+1] = 0;
            }
            if(tid==1) {
                mp_mask[0][0] = 0;
                mp_mask[0][MEGAPIXEL_SIZE+1] = 0;
                mp_mask[MEGAPIXEL_SIZE+1][0] = 0;
                mp_mask[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE+1] = 0;
            }

            // Aqui podemos realizar a computação na memória compartilhada
            // Synchronização para garantia de consistência
            __syncthreads();

            // Análogo a propagação SR
            // auxStart = clock64();
            do{
                elementsUpdated = XForwardPropagation(mp_marker, mp_mask);
                elementsUpdated = YDownwardPropagation(mp_marker, mp_mask) or elementsUpdated;
                elementsUpdated = XBackwardPropagation(mp_marker, mp_mask) or elementsUpdated;
                elementsUpdated = YUpwardPropagation(mp_marker, mp_mask) or elementsUpdated;
            }while(elementsUpdated);
            // MP_Processing += clock64() - auxStart;


            // Escrevemos o resultado de volta na memória global
            // Bloco principal
            // auxStart = clock64();
            countElementsUpdated = 0;
            #pragma unroll
            for(i=0; i<MEGAPIXEL_SIZE; i+=4) {
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + i + rowOffset) * pitchMarker);

                // Carregamos o conteudo atual da memória global
                val = *((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])));
                do{
                    elementsUpdated = 0;
                    assumed = val;
                    // Verificamos se alguma alteração foi feita
                    // Note que um int comporta 4 unsigned char
                    for(j=0; j<4; j++) {
                        if(mp_marker[i+rowOffset+1][wordOffset +j+1] > valAsChar[j]) {
                            valAsChar[j] = mp_marker[i+rowOffset+1][wordOffset +j+1];
                            elementsUpdated = 1;
                        }
                    }

                    // Saimos do loop caso nenhum elemento dessa palavra precise ser atualizado 
                    if(!elementsUpdated) {
                        break;
                    }

                    // Realizamos a escrita na memória global
                    val = atomicCAS((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])), assumed, val);
                } while(val != assumed);
                countElementsUpdated += elementsUpdated;
            }


            // Verificamos se esse megapixel deve ser reinserido na fila de megapixels
            countElementsUpdated = __syncthreads_or(countElementsUpdated);
            if(tid==0 and countElementsUpdated) {
                writeBuffer[writeSize++] = readBuffer[currentRegister];
            }

            // Lateral Superior
            // Resta a flag de quem nao entra no loop
            elementsUpdated = 0;
            if(tid<8 and readBuffer[currentRegister].x > 0) {
                // Olhamos para a última linha do bloco de cima
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x - 1) * pitchMarker);

                // Carregamos o conteudo atual da memória global
                val = *((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])));
                do{
                    elementsUpdated = 0;
                    assumed = val;
                    // Verificamos se alguma alteração foi feita
                    // Note que um int comporta 4 unsigned char
                    for(j=0; j<4; j++) {
                        if(mp_marker[0][wordOffset +j+1] > valAsChar[j]) {
                            valAsChar[j] = mp_marker[0][wordOffset +j+1];
                            elementsUpdated = 1;
                        }
                    }

                    // Saimos do loop caso nenhum elemento dessa palavra precise ser atualizado 
                    if(!elementsUpdated) {
                        break;
                    }

                    // Realizamos a escrita na memória global
                    val = atomicCAS((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])), assumed, val);
                } while(val != assumed);

            }


            // ------------collective verification
            countElementsUpdated = __syncthreads_or(elementsUpdated);
            if(tid==0 and countElementsUpdated) {
                writeBuffer[writeSize++] = readBuffer[currentRegister];
                writeBuffer[writeSize - 1].x -= 1;
                // globalQueueWriteBuffer[totalToWrite++] = make_int2(globalQueueReadBuffer.x-1, globalQueueReadBuffer.y);
            }

                  
            // Lateral inferior
            // Reseta a flag de quem nao entra no loop
            elementsUpdated = 0;
            if(tid<8 and readBuffer[currentRegister].x < (size/MEGAPIXEL_SIZE)-1) {
                // Olhamos para a primeira linha do bloco de baixo
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*(readBuffer[currentRegister].x+1)) * pitchMarker);

                // Carregamos o conteudo atual da memória global
                val = *((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])));
                do{
                    elementsUpdated = 0;
                    assumed = val;
                    // Verificamos se alguma alteração foi feita
                    // Note que um int comporta 4 unsigned char
                    for(j=0; j<4; j++) {
                        if(mp_marker[MEGAPIXEL_SIZE+1][wordOffset +j+1] > valAsChar[j]) {
                            valAsChar[j] = mp_marker[MEGAPIXEL_SIZE+1][wordOffset +j+1];
                            elementsUpdated = 1;
                        }
                    }

                    // Saimos do loop caso nenhum elemento dessa palavra precise ser atualizado 
                    if(!elementsUpdated) {
                        break;
                    }

                    // Realizamos a escrita na memória global
                    val = atomicCAS((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE + wordOffset])), assumed, val);
                } while(val != assumed);

            }

            // ------------collective verification
            countElementsUpdated = __syncthreads_or(elementsUpdated);

            if(tid==0 and countElementsUpdated) {
                writeBuffer[writeSize++] = readBuffer[currentRegister];
                writeBuffer[writeSize - 1].x += 1;
                // globalQueueWriteBuffer[totalToWrite++] = make_int2(globalQueueReadBuffer.x+1, globalQueueReadBuffer.y);
            }

            // // Laterais esquerda e direita
            // Lateral esquerda
            if(readBuffer[currentRegister].y > 0) {
                // Cada thread olha para a sua linha no megapixel
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMarker);
                
                // Olhamos para o megapixel à esquera
                // Novamente trabalhamos com 4 bytes
                val = *((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE - 4])));
                do {
                    elementsUpdated = 0;
                    assumed = val;
                    // Verificamos se alguma alteração foi feita
                    // Note que um int comporta 4 unsigned char
                    if(mp_marker[tid+1][0] > valAsChar[3]) {
                        valAsChar[3] = mp_marker[tid+1][0];
                        elementsUpdated = 1;
                    }
                    else {
                        break;
                    }

                    // Realizamos a escrita na memória global
                    val = atomicCAS((unsigned int*)(&(markerRow[readBuffer[currentRegister].y*MEGAPIXEL_SIZE - 4])), assumed, val);
                } while(assumed != val);

                // ------------collective verification
                countElementsUpdated = __syncthreads_or(elementsUpdated);

                if(tid==0 and countElementsUpdated) {
                    writeBuffer[writeSize++] = readBuffer[currentRegister];
                    writeBuffer[writeSize - 1].y -= 1;
                    // globalQueueWriteBuffer[totalToWrite++] = make_int2(globalQueueReadBuffer.x, globalQueueReadBuffer.y-1);
                }
            }

            // Lateral direita
            if(readBuffer[currentRegister].y < (size/MEGAPIXEL_SIZE)-1) {
                // Cada thread olha para a sua linha no megapixel
                markerRow = (volatile unsigned char*)((char*)marker + (MEGAPIXEL_SIZE*readBuffer[currentRegister].x + tid) * pitchMarker);
                
                // Olhamos para o megapixel à esquera
                // Novamente trabalhamos com 4 bytes
                val = *((unsigned int*)(&(markerRow[(readBuffer[currentRegister].y+1)*MEGAPIXEL_SIZE])));
                do {
                    elementsUpdated = 0;
                    assumed = val;
                    // Verificamos se alguma alteração foi feita
                    // Note que um int comporta 4 unsigned char
                    if(mp_marker[tid+1][MEGAPIXEL_SIZE+1] > valAsChar[0]) {
                        valAsChar[0] = mp_marker[tid+1][MEGAPIXEL_SIZE+1];
                        elementsUpdated = 1;
                    }
                    else {
                        break;
                    }

                    // Realizamos a escrita na memória global
                    val = atomicCAS((unsigned int*)(&(markerRow[(readBuffer[currentRegister].y+1)*MEGAPIXEL_SIZE])), assumed, val);
                } while(assumed != val);

                // ------------collective verification
                countElementsUpdated = __syncthreads_or(elementsUpdated);

                if(tid==0 and countElementsUpdated) {
                    writeBuffer[writeSize++] = readBuffer[currentRegister];
                    writeBuffer[writeSize - 1].y += 1;
                    // globalQueueWriteBuffer[totalToWrite++] = make_int2(globalQueueReadBuffer.x, globalQueueReadBuffer.y+1);
                }
            }

            // Para garantir a consistência da memória compartilhada
            __syncthreads();
            // MP_Dump += clock64() - auxStart;


            // Verificamos se devemos escrever o buffer de escrita na memória global
            // auxStart = clock64();
            if(BUFFER_SIZE - 5 < writeSize) {
                // Aqui escrevemos o buffer na memória global
                writtenAmount = insertIntoGlobalQueue(gq->writeQueue, &(gq->writeQueueSize), GLOBAL_QUEUE_SIZE, writeBuffer, writeSize);

                // Restaurando a fila
                if(tid==0) {
                    // Verificamos se a flag de overflow deve ser settada
                    if(writtenAmount < writeSize) {
                        *memLeak = 1;
                    }
                    writeSize = 0;
                }
            }
            // queueWrite += clock64() - auxStart;

            // Esperamos todo mundo encerrar o loop
            // ------------NÃO SEI SE PRECISA DESSA SINCRONIZAÇÃO
            __syncthreads();
        }

        // auxStart = clock64();
    }

    // Aqui escrevemos na memória global os resultados do buffer que ainda não foram escritos
    // auxStart = clock64();
    if(writeSize > 0) {
        writtenAmount = insertIntoGlobalQueue(gq->writeQueue, &(gq->writeQueueSize), GLOBAL_QUEUE_SIZE, writeBuffer, writeSize);
        
        // Verificamos se a flag de overflow deve ser settada
        if(tid==0 and writtenAmount < writeSize) {
            *memLeak = 1;
        }
    }
    // queueWrite += clock64() - auxStart;

    // if(tid==0) {
    //     atomicAdd(&times[QUEUE_READ], queueRead);
    //     atomicAdd(&times[MP_FETCH], MP_Fetch);
    //     atomicAdd(&times[MP_PROCESSING], MP_Processing);
    //     atomicAdd(&times[MP_DUMP], MP_Dump);
    //     atomicAdd(&times[QUEUE_WRITE], queueWrite);
    // }

}

// Propagação da imagem inteira
// Assume que a imagem é quadrada e que o tamanho da imagem é múltiplo de 32
// Assume que os blocos tem 32 threads
void __global__ XForwardPropagationEntireImage(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask, int size) {
    const int baseRow = blockIdx.x * MEGAPIXEL_SIZE;
    unsigned char* markerRow;
    unsigned char* maskRow;

    int tid = threadIdx.x;

    // Mega pixel
    __shared__ unsigned char mp_marker[MEGAPIXEL_SIZE][MEGAPIXEL_SIZE+1];
    __shared__ unsigned char mp_mask[MEGAPIXEL_SIZE][MEGAPIXEL_SIZE+1];

    int col, i;

    // inicializamos a coluna de carry do megapixel
    mp_marker[tid][0] = 0;

    for(col=0; col<size; col+=MEGAPIXEL_SIZE) {
        // Realizamos a leitura do megapixel
        // Linha por linha
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (baseRow + i)*pitchMarker);
            maskRow = (unsigned char*)((char*)mask + (baseRow + i)*pitchMask);
            mp_marker[i][tid+1] = markerRow[col + tid];
            mp_mask[i][tid+1] = maskRow[col + tid];
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Realizamos a propagação, cada thread pega uma linha
        // Note que como essa rotina vai ser executada apenas uma vez, podemos trocar uma comparação por uma escrita
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            if(mp_marker[tid][i] > mp_marker[tid][i+1]) {
                mp_marker[tid][i+1] = min(mp_marker[tid][i], mp_mask[tid][i+1]);
            }
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Escrevemos o resultado de volta na memória global
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (baseRow + i)*pitchMarker);
            markerRow[col + tid] = mp_marker[i][tid+1];
        }

        // Copiamos a coluna de carry para continuar a propagação
        mp_marker[tid][0] = mp_marker[tid][MEGAPIXEL_SIZE];
    }
}

// Assume que a imagem é quadrada e que o tamanho da imagem é múltiplo de 32
// Assume que os blocos tem 32 threads
void __global__ XBackwardPropagationEntireImage(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask, int size) {
    const int baseRow = blockIdx.x * MEGAPIXEL_SIZE;
    unsigned char* markerRow;
    unsigned char* maskRow;

    int tid = threadIdx.x;

    // Mega pixel
    __shared__ unsigned char mp_marker[MEGAPIXEL_SIZE][MEGAPIXEL_SIZE+1];
    __shared__ unsigned char mp_mask[MEGAPIXEL_SIZE][MEGAPIXEL_SIZE+1];

    int col, i;

    // inicializamos a coluna de carry do megapixel
    mp_marker[tid][MEGAPIXEL_SIZE] = 0;

    for(col=size-MEGAPIXEL_SIZE; col>=0; col-=MEGAPIXEL_SIZE) {
        // Realizamos a leitura do megapixel
        // Linha por linha
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (baseRow + i)*pitchMarker);
            maskRow = (unsigned char*)((char*)mask + (baseRow + i)*pitchMask);
            mp_marker[i][tid] = markerRow[col + tid];
            mp_mask[i][tid] = maskRow[col + tid];
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Realizamos a propagação, cada thread pega uma linha
        // Note que como essa rotina vai ser executada apenas uma vez, podemos trocar uma comparação por uma escrita
        for(i=MEGAPIXEL_SIZE; i>0; i--) {
            if(mp_marker[tid][i] > mp_marker[tid][i-1]) {
                mp_marker[tid][i-1] = min(mp_marker[tid][i], mp_mask[tid][i-1]);
            }
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Escrevemos o resultado de volta na memória global
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (baseRow + i)*pitchMarker);
            markerRow[col + tid] = mp_marker[i][tid];
        }

        // Copiamos a coluna de carry para continuar a propagação
        mp_marker[tid][MEGAPIXEL_SIZE] = mp_marker[tid][0];
    }
}

// Assume que a imagem é quadrada e que o tamanho da imagem é múltiplo de 32
// Assume que os blocos tem 32 threads
void __global__ YUpwardPropagationEntireImage(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask, int size) {
    const int baseCol = blockIdx.y * MEGAPIXEL_SIZE;
    unsigned char* markerRow;
    unsigned char* maskRow;

    int tid = threadIdx.x;

    // Mega pixel
    __shared__ unsigned char mp_marker[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE];
    __shared__ unsigned char mp_mask[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE];

    int row, i;

    // inicializamos a coluna de carry do megapixel
    mp_marker[MEGAPIXEL_SIZE][tid] = 0;

    for(row=size-MEGAPIXEL_SIZE; row>=0; row-=MEGAPIXEL_SIZE) {
        // Realizamos a leitura do megapixel
        // Linha por linha
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (row + i)*pitchMarker);
            maskRow = (unsigned char*)((char*)mask + (row + i)*pitchMask);
            mp_marker[i][tid] = markerRow[baseCol + tid];
            mp_mask[i][tid] = maskRow[baseCol + tid];
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Realizamos a propagação, cada thread pega uma linha
        // Note que como essa rotina vai ser executada apenas uma vez, podemos trocar uma comparação por uma escrita
        for(i=MEGAPIXEL_SIZE; i>0; i--) {
            if(mp_marker[i][tid] > mp_marker[i-1][tid]) {
                mp_marker[i-1][tid] = min(mp_marker[i][tid], mp_mask[i-1][tid]);
            }
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Escrevemos o resultado de volta na memória global
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (row + i)*pitchMarker);
            markerRow[baseCol + tid] = mp_marker[i][tid];
        }

        // Copiamos a coluna de carry para continuar a propagação
        mp_marker[MEGAPIXEL_SIZE][tid] = mp_marker[0][tid];
    }
}

void __global__ YDownwardPropagationEntireImage(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask, int size) {
    const int baseCol = blockIdx.y * MEGAPIXEL_SIZE;
    unsigned char* markerRow;
    unsigned char* maskRow;

    int tid = threadIdx.x;

    // Mega pixel
    __shared__ unsigned char mp_marker[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE];
    __shared__ unsigned char mp_mask[MEGAPIXEL_SIZE+1][MEGAPIXEL_SIZE];

    int row, i;

    // inicializamos a coluna de carry do megapixel
    mp_marker[0][tid] = 0;

    for(row=0; row<size; row+=MEGAPIXEL_SIZE) {
        // Realizamos a leitura do megapixel
        // Linha por linha
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (row + i)*pitchMarker);
            maskRow = (unsigned char*)((char*)mask + (row + i)*pitchMask);
            mp_marker[i+1][tid] = markerRow[baseCol + tid];
            mp_mask[i+1][tid] = maskRow[baseCol + tid];
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Realizamos a propagação, cada thread pega uma linha
        // Note que como essa rotina vai ser executada apenas uma vez, podemos trocar uma comparação por uma escrita
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            if(mp_marker[i][tid] > mp_marker[i+1][tid]) {
                mp_marker[i+1][tid] = min(mp_marker[i][tid], mp_mask[i+1][tid]);
            }
        }

        // Sincronizamos para garantia de cosnsitência da memória compartilhada
        __syncthreads();

        // Escrevemos o resultado de volta na memória global
        for(i=0; i<MEGAPIXEL_SIZE; i++) {
            markerRow = (unsigned char*)((char*)marker + (row + i)*pitchMarker);
            markerRow[baseCol + tid] = mp_marker[i+1][tid];
        }

        // Copiamos a coluna de carry para continuar a propagação
        mp_marker[0][tid] = mp_marker[MEGAPIXEL_SIZE][tid];
    }
}

// ---------------------------------------------------------------------
// RASTER ANTI-RASTER SCAN ALTERNATIVO
// ---------------------------------------------------------------------

template <typename T>
__global__ void
iRec1DForward_X_dilation ( T* marker, const T* mask, const unsigned int sx, const unsigned int sy, bool* change )
{
	const unsigned int x = (threadIdx.x + threadIdx.y * XX_THREADS) % WARP_SIZE;
	const unsigned int y = (threadIdx.x + threadIdx.y * XX_THREADS) / WARP_SIZE;
	const unsigned int ychunk = WARP_SIZE / XX_THREADS;
	const unsigned int xstop = sx - WARP_SIZE;
//	printf("(tx, ty) -> (x, y) : (%d, %d)->(%d,%d)\n", threadIdx.x, threadIdx.y, x, y);

	// XY_THREADS should be 32==warpSize, XX_THREADS should be 4 or 8.
	// init to 0...
	volatile __shared__ T s_marker[XY_THREADS][WARP_SIZE+1];
	volatile __shared__ T s_mask  [XY_THREADS][WARP_SIZE+1];
	volatile unsigned int s_change = 0;
	T s_old, s_new;
	unsigned int startx;
	unsigned int start;



	s_marker[threadIdx.y][WARP_SIZE] = 0;  // only need x=0 to be 0

	// the increment allows overlap by 1 between iterations to move the data to next block.
	for (startx = 0; startx < xstop; startx += WARP_SIZE) {
		start = (blockIdx.x * XY_THREADS + y * ychunk) * sx + startx + x;
//			printf("tx: %d, ty: %d, x: %d, y: %d, startx: %d, start: %d", threadIdx.x, threadIdx.y, x, y, startx, start);

		s_marker[threadIdx.y][0] = s_marker[threadIdx.y][WARP_SIZE];

		// copy part of marker and mask to shared memory.  works for 1 warp at a time...
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			s_marker[y * ychunk+i][x+1] = marker[start + i*sx];
			s_mask  [y * ychunk+i][x+1] = mask[start + i*sx];
		}

		// perform iteration   all X threads do the same operations, so there may be read/write hazards.  but the output is the same.
		// this is looping for BLOCK_SIZE times, and each iteration the final results are propagated 1 step closer to tx.
//			if (threadIdx.x == 0) {  // have all threads do the same work
    //#pragma unroll
    if (threadIdx.y + blockIdx.x * XY_THREADS < sy) {   //require dimension to be perfectly padded.
            for (unsigned int i = 1; i <= WARP_SIZE; ++i) {
                s_old = s_marker[threadIdx.y][i];
                s_new = min( max( s_marker[threadIdx.y][i-1], s_old ), s_mask[threadIdx.y][i] );
                s_change |= s_new ^ s_old;
                s_marker[threadIdx.y][i] = s_new;
            }
    }
		// output result back to global memory and set up for next x chunk
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			marker[start + i*sx] = s_marker[y * ychunk+i][x+1];
		}
//			printf("startx: %d, change = %d\n", startx, s_change);

	}

	if (startx < sx) {
		s_marker[threadIdx.y][0] = s_marker[threadIdx.y][sx-startx];  // getting ix-1st entry, which has been offsetted by 1 in s_marker
		// shared mem copy
		startx = sx - WARP_SIZE;
		start = (blockIdx.x * XY_THREADS + y * ychunk) * sx + startx + x;
//			printf("tx: %d, ty: %d, x: %d, y: %d, startx: %d, start: %d", threadIdx.x, threadIdx.y, x, y, startx, start);

		// copy part of marker and mask to shared memory.  works for 1 warp at a time...
    //#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			s_marker[y * ychunk+i][x+1] = marker[start + i*sx];
			s_mask  [y * ychunk+i][x+1] = mask[start + i*sx];
		}

		// perform iteration   all X threads do the same operations, so there may be read/write hazards.  but the output is the same.
		// this is looping for BLOCK_SIZE times, and each iteration the final results are propagated 1 step closer to tx.
    //#pragma unroll
    if (threadIdx.y + blockIdx.x * XY_THREADS < sy) {   //require dimension to be perfectly padded.
            for (unsigned int i = 1; i <= WARP_SIZE; ++i) {
                s_old = s_marker[threadIdx.y][i];
                s_new = min( max( s_marker[threadIdx.y][i-1], s_old ), s_mask[threadIdx.y][i] );
                s_change |= s_new ^ s_old;
                s_marker[threadIdx.y][i] = s_new;
            }
    }
		// output result back to global memory and set up for next x chunk
//#pragma unroll
    for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			marker[start + i*sx] = s_marker[y * ychunk+i][x+1];
		}
	}


//	__syncthreads();
	if (s_change > 0) *change = true;
//	__syncthreads();

}

template <typename T>
__global__ void
iRec1DForward_Y_dilation ( T* __restrict__ marker, const T* __restrict__ mask, const int sx, const int sy, bool* __restrict__ change )
{
	// parallelize along x.
	const int tx = threadIdx.x;
	const int bx = blockIdx.x * MAX_THREADS;

	unsigned int  s_change = 0;
	T s_old, s_new, s_prev;
	
if ( (bx + tx) < sx ) {

		s_prev = 0;

		for (int iy = 0; iy < sy; ++iy) {
			// copy part of marker and mask to shared memory
			s_old = marker[iy * sx + bx + tx];

			// perform iteration
			s_new = min( max( s_prev, s_old ), mask[iy * sx + bx + tx] );
			s_change |= s_old ^ s_new;
			s_prev = s_new;

			// output result back to global memory
			marker[iy * sx + bx + tx] = s_new;

		}
}
		
		if (s_change != 0) *change = true;


}

template <typename T>
__global__ void
iRec1DBackward_X_dilation ( T* __restrict__ marker, const T* __restrict__ mask, const int sx, const int sy, bool* __restrict__ change )
{

	const unsigned int x = (threadIdx.x + threadIdx.y * XX_THREADS) % WARP_SIZE;
	const unsigned int y = (threadIdx.x + threadIdx.y * XX_THREADS) / WARP_SIZE;
	const unsigned int ychunk = WARP_SIZE / XX_THREADS;
	const unsigned int xstop = sx - WARP_SIZE;
	//	printf("(tx, ty) -> (x, y) : (%d, %d)->(%d,%d)\n", threadIdx.x, threadIdx.y, x, y);

	// XY_THREADS should be 32==warpSize, XX_THREADS should be 4 or 8.
	// init to 0...
	volatile __shared__ T s_marker[XY_THREADS][WARP_SIZE+1];
	volatile __shared__ T s_mask  [XY_THREADS][WARP_SIZE+1];
	volatile unsigned int s_change = 0;
	T s_old, s_new;
	int startx;
	unsigned int start;
	
	s_marker[threadIdx.y][0] = 0;  // only need x=WARPSIZE to be 0

	// the increment allows overlap by 1 between iterations to move the data to next block.
	for (startx = xstop; startx > 0; startx -= WARP_SIZE) {
		start = (blockIdx.x * XY_THREADS + y * ychunk) * sx + startx + x;
//			printf("tx: %d, ty: %d, x: %d, y: %d, startx: %d, start: %d", threadIdx.x, threadIdx.y, x, y, startx, start);

		s_marker[threadIdx.y][WARP_SIZE] = s_marker[threadIdx.y][0];

		// copy part of marker and mask to shared memory.  works for 1 warp at a time...
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			s_marker[y * ychunk+i][x] = marker[start + i*sx];
			s_mask  [y * ychunk+i][x] = mask[start + i*sx];
		}

		// perform iteration   all X threads do the same operations, so there may be read/write hazards.  but the output is the same.
		// this is looping for BLOCK_SIZE times, and each iteration the final results are propagated 1 step closer to tx.
//			if (threadIdx.x == 0) {  // have all threads do the same work
//#pragma unroll
        if (threadIdx.y + blockIdx.x * XY_THREADS < sy) {   //require dimension to be perfectly padded.
                for (int i = WARP_SIZE - 1; i >= 0; --i) {
                    s_old = s_marker[threadIdx.y][i];
                    s_new = min( max( s_marker[threadIdx.y][i+1], s_old ), s_mask[threadIdx.y][i] );
                    s_change |= s_new ^ s_old;
                    s_marker[threadIdx.y][i] = s_new;
                }
        }
		// output result back to global memory and set up for next x chunk
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			marker[start + i*sx] = s_marker[y * ychunk+i][x];
		}
//			printf("startx: %d, change = %d\n", startx, s_change);
	}

	if (startx <= 0) {
		s_marker[threadIdx.y][WARP_SIZE] = s_marker[threadIdx.y][-startx];  // getting ix-1st entry, which has been offsetted by 1 in s_marker
		// shared mem copy
		startx = 0;
		start = (blockIdx.x * XY_THREADS + y * ychunk) * sx + startx + x;
//			printf("tx: %d, ty: %d, x: %d, y: %d, startx: %d, start: %d", threadIdx.x, threadIdx.y, x, y, startx, start);

		// copy part of marker and mask to shared memory.  works for 1 warp at a time...
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			s_marker[y * ychunk+i][x] = marker[start + i*sx];
			s_mask  [y * ychunk+i][x] = mask[start + i*sx];
		}

		// perform iteration   all X threads do the same operations, so there may be read/write hazards.  but the output is the same.
		// this is looping for BLOCK_SIZE times, and each iteration the final results are propagated 1 step closer to tx.
//#pragma unroll
        if (threadIdx.y + blockIdx.x * XY_THREADS < sy) {   //require dimension to be perfectly padded.
                for (int i = WARP_SIZE - 1; i >= 0; --i) {
                    s_old = s_marker[threadIdx.y][i];
                    s_new = min( max( s_marker[threadIdx.y][i+1], s_old ), s_mask[threadIdx.y][i] );
                    s_change |= s_new ^ s_old;
                    s_marker[threadIdx.y][i] = s_new;
                }
        }
		// output result back to global memory and set up for next x chunk
//#pragma unroll
		for (unsigned int i = 0; i < ychunk && y*ychunk+i < sy; ++i) {
			marker[start + i*sx] = s_marker[y * ychunk+i][x];
		}
	}

	//	__syncthreads();
	if (s_change > 0) *change = true;
	//	__syncthreads();
}

template <typename T>
	__global__ void
iRec1DBackward_Y_dilation ( T* __restrict__ marker, const T* __restrict__ mask, const unsigned int sx, const unsigned int sy, bool* __restrict__ change )
{

	const int tx = threadIdx.x;
	const int bx = blockIdx.x * MAX_THREADS;

	unsigned int s_change=0;
	T s_old, s_new, s_prev;

	if ( (bx + tx) < sx ) {

		s_prev = 0;

		for (int iy = sy - 1; iy >= 0; --iy) {

			// copy part of marker and mask to shared memory
			s_old = marker[iy * sx + bx + tx];

			// perform iteration
			s_new = min( max( s_prev, s_old ), mask[iy * sx + bx + tx] );
			s_change |= s_old ^ s_new;
			s_prev = s_new;

			// output result back to global memory
			marker[iy * sx + bx + tx] = s_new;
		}
	}

	if (s_change != 0) *change = true;

}

// Assumimos que a imagem é quadrada
void one_alternative_raster_pass(unsigned char* marker, unsigned char* mask, int size) {
    // Dimensões de Launch
    dim3 threadsx( XX_THREADS, XY_THREADS );
	dim3 blocksx( (size + threadsx.y - 1) / threadsx.y );
    dim3 threadsy( MAX_THREADS );
	dim3 blocksy( (size + threadsy.x - 1) / threadsy.x );

    bool *d_change;

    int i;
    hipEvent_t start[4], stop[4];
    float time[4];

    for(i=0; i<4; i++) {
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
    }


    // Alocamos a variável
    hipMalloc(&d_change, sizeof(bool));

    // dopredny pruchod pres osu X
    hipEventRecord(start[0]);
    iRec1DForward_X_dilation <<< blocksx, threadsx, 0, 0 >>> ( marker, mask, size, size, d_change );
    hipEventRecord(stop[0]);

    // dopredny pruchod pres osu Y
    hipEventRecord(start[1]);
    iRec1DForward_Y_dilation <<< blocksy, threadsy, 0, 0 >>> ( marker, mask, size, size, d_change );
    hipEventRecord(stop[1]);

    // zpetny pruchod pres osu X
    hipEventRecord(start[2]);
    iRec1DBackward_X_dilation<<< blocksx, threadsx, 0, 0 >>> ( marker, mask, size, size, d_change );
    hipEventRecord(stop[2]);

    // zpetny pruchod pres osu Y
    hipEventRecord(start[3]);
    iRec1DBackward_Y_dilation<<< blocksy, threadsy, 0, 0 >>> ( marker, mask, size, size, d_change );
    hipEventRecord(stop[3]);

    hipDeviceSynchronize();

    for(i=0; i<4; i++) {
        hipEventElapsedTime(&time[i], start[i], stop[i]);
        //printf("Passada %d: %f (ms)\n", i+1, time[i]);
        printf("%f\n", time[i]);
    }

    hipFree(d_change);
    for(i=0; i<4; i++) {
        hipEventDestroy(start[i]);
        hipEventDestroy(stop[i]);
    }

}


void one_current_pass(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask, int size) {
    int i;
    hipEvent_t start[4], stop[4];
    float time[4];

    for(i=0; i<4; i++) {
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
    }

    // dopredny pruchod pres osu X
    hipEventRecord(start[0]);
    XForwardPropagationEntireImage<<<dim3(size/MEGAPIXEL_SIZE, 1), 32>>>(marker, mask, pitchMarker, pitchMask, size);
    hipEventRecord(stop[0]);

    // dopredny pruchod pres osu Y
    hipEventRecord(start[1]);
    YDownwardPropagationEntireImage<<<dim3(1, size/MEGAPIXEL_SIZE), 32>>>(marker, mask, pitchMarker, pitchMask, size);
    hipEventRecord(stop[1]);

    // zpetny pruchod pres osu X
    hipEventRecord(start[2]);
    XBackwardPropagationEntireImage<<<dim3(size/MEGAPIXEL_SIZE, 1), 32>>>(marker, mask, pitchMarker, pitchMask, size);
    hipEventRecord(stop[2]);

    // zpetny pruchod pres osu Y
    hipEventRecord(start[3]);
    YUpwardPropagationEntireImage<<<dim3(1, size/MEGAPIXEL_SIZE), 32>>>(marker, mask, pitchMarker, pitchMask, size);
    hipEventRecord(stop[3]);

    hipDeviceSynchronize();

    for(i=0; i<4; i++) {
        hipEventElapsedTime(&time[i], start[i], stop[i]);
        //printf("Passada %d: %f (ms)\n", i+1, time[i]);
        printf("%f\n", time[i]);
    }

    for(i=0; i<4; i++) {
        hipEventDestroy(start[i]);
        hipEventDestroy(stop[i]);
    }
}

// ---------------------------------------------------------------------


void MorphologicalReconstruction(unsigned char* marker, unsigned char* mask, 
            size_t pitchMarker, size_t pitchMask, int size)
{
    int blockSize = 32;
    int maximumBlockOccupancy = 2048;
    int gridSize = maximumBlockOccupancy;
    
    // dim3 threadsx( XX_THREADS, XY_THREADS );    
    // dim3 blocksx( (size + threadsx.y - 1) / threadsx.y );
    // dim3 threadsy( MAX_THREADS );
    // dim3 blocksy( (size + threadsy.x - 1) / threadsy.x );

    // bool *d_change;

    // unsigned long long *d_times;
    // hipMalloc(&d_times, 5*sizeof(unsigned long long));

    // unsigned long long h_times[5];

    // float morphReconTime = 0;
    // float queueMaintainceTime = 0;

    // hipEvent_t start, stop;

    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    int megapixelGridSize = size/MEGAPIXEL_SIZE;
    
    GlobalQueue *gq;
    // Assumimos que a imagem é quadrada
    int readQueueSize;

    int hMemLeak;
    int* dMemLeak;

    unsigned long long megapixelsProcessed = 0;

    //int *d_countIters;
    //int *d_maxIters;
    //int h_countIters;
    //int h_maxIters;

    //hipMalloc(&d_countIters, sizeof(int));
    //hipMalloc(&d_maxIters, sizeof(int));

    // inicializa o mecanismo de controle de overflow
    hipMalloc(&dMemLeak, sizeof(int)); 

    // Inicializa a struct da fila global
    hipMalloc(&gq, sizeof(GlobalQueue));

    // Alocamos a variável usada no raster anti raster alternativo
    //hipMalloc(&d_change, sizeof(bool));

    // Inicializa os atributos da struct 
    initGlobalQueue(gq);

    // auto tStart = std::chrono::high_resolution_clock::now();
    do{
        // Inicializa a fila de megapixels e a flag de overflow de memória
        hipMemset(dMemLeak, 0, sizeof(int));
        initFillGlobalQueue<<<dim3(megapixelGridSize/FILL_BLOCK_SIZE,megapixelGridSize/FILL_BLOCK_SIZE), 
                    dim3(FILL_BLOCK_SIZE, FILL_BLOCK_SIZE)>>>(gq);

        readQueueSize = megapixelGridSize*megapixelGridSize;
        // Varreduras raster e antiraster
        // XForwardPropagationEntireImage<<<dim3(size/MEGAPIXEL_SIZE, 1), 32>>>(marker, mask, pitchMarker, pitchMask, size);
        // YDownwardPropagationEntireImage<<<dim3(1, size/MEGAPIXEL_SIZE), 32>>>(marker, mask, pitchMarker, pitchMask, size);
        // XBackwardPropagationEntireImage<<<dim3(size/MEGAPIXEL_SIZE, 1), 32>>>(marker, mask, pitchMarker, pitchMask, size);
        // YUpwardPropagationEntireImage<<<dim3(1, size/MEGAPIXEL_SIZE), 32>>>(marker, mask, pitchMarker, pitchMask, size);

        // dopredny pruchod pres osu X
        //iRec1DForward_X_dilation <<< blocksx, threadsx, 0, 0 >>> ( marker, mask, size, size, d_change );
        // dopredny pruchod pres osu Y
        //iRec1DForward_Y_dilation <<< blocksy, threadsy, 0, 0 >>> ( marker, mask, size, size, d_change );
        // zpetny pruchod pres osu X
        //iRec1DBackward_X_dilation<<< blocksx, threadsx, 0, 0 >>> ( marker, mask, size, size, d_change );
        // zpetny pruchod pres osu Y
        //iRec1DBackward_Y_dilation<<< blocksy, threadsy, 0, 0 >>> ( marker, mask, size, size, d_change );
        //gridSize = 1344;
        
        // Computamos a reconstrução morfológica
        while(readQueueSize > 0) {
            //printf("%d\n", readQueueSize);
            //gridSize = min(readQueueSize/16, 1344);
            // Caso existam menos que 16 registros na fila
            //if(gridSize==0) {
            //    gridSize++;
            //}
            gridSize = min(readQueueSize, maximumBlockOccupancy);
            //hipMemset(d_countIters, 0, sizeof(int));
            //hipMemset(d_maxIters, 0, sizeof(int));
            
            //hipEventRecord(start, 0);
            // Kernel que consome a fila de leitura
            // hipEventRecord(start);
            megapixelsProcessed += readQueueSize;
            MorphologicalReconstructionKernel<<<gridSize, blockSize>>>(gq, marker, mask, pitchMarker, pitchMask, size, dMemLeak);
            // hipEventRecord(stop);
            // hipEventSynchronize(stop);
            // hipEventElapsedTime(&morphReconTime, start, stop);

            //hipEventRecord(stop, 0);

            // hipEventRecord(start);
            swapQueues<<<1,1>>>(gq);
            // hipEventRecord(stop);
            // hipEventSynchronize(stop);
            // hipEventElapsedTime(&queueMaintainceTime, start, stop);
            
            // hipDeviceSynchronize();

            //hipEventElapsedTime(&time, start, stop);
            //hipMemcpy(&h_countIters, d_countIters, sizeof(int), hipMemcpyDeviceToHost);
            //hipMemcpy(&h_maxIters, d_maxIters, sizeof(int), hipMemcpyDeviceToHost);
            //printf("Queue size: %d\n", readQueueSize);
            // Fazer mais testes depois---------------------------------------------
            // printf("%d\t%f\t%f\t%f\t%f\n", readQueueSize, morphReconTime, queueMaintainceTime, 0.0, 0.0);
            hipMemcpy(&readQueueSize, &(gq->readQueueSize), sizeof(int), hipMemcpyDeviceToHost);

            // Nao faz sentido processar 64K megapixels se a imagem so�tem 16KB
            //if(readQueueSize > megapixelGridSize*megapixelGridSize) {
            //  initFillGlobalQueue<<<dim3(megapixelGridSize/FILL_BLOCK_SIZE,megapixelGridSize/FILL_BLOCK_SIZE), 
            //        dim3(FILL_BLOCK_SIZE, FILL_BLOCK_SIZE)>>>(gq);
            //  readQueueSize = megapixelGridSize*megapixelGridSize;
            //}

            //hipDeviceSynchronize();
        }

        // Dispara o kernel de computação da reconstrução morfológica
        // MorphologicalReconstructionKernel<<<gridSize, blockSize>>>(gq, marker, mask, pitchMarker, pitchMask, size, dMemLeak);
        // hipDeviceSynchronize();

        // Obtem a flag de overflow
        hipMemcpy(&hMemLeak, dMemLeak, sizeof(int), hipMemcpyDeviceToHost);
        // hipDeviceSynchronize();

        printf("MemLeak: %d\n", hMemLeak);
    } while(hMemLeak);

    // auto tEnd = std::chrono::high_resolution_clock::now();

    // auto ms_int = std::chrono::duration_cast<std::chrono::milliseconds>(tEnd - tStart);
    // printf("Tempo de loop: %ld (ms)\n", ms_int.count());

    //printf("Processed Mps: %llu\n", megapixelsProcessed);

    // Libera a memória alocada
    freeGlobalQueue<<<1,1>>>(gq);
    hipDeviceSynchronize();

    // hipMemcpy(h_times, d_times, 5*sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    /*printf("Queue Read: %llu\n", h_times[QUEUE_READ]);
    printf("MP Fetch: %llu\n", h_times[MP_FETCH]);
    printf("MP Processing: %llu\n", h_times[MP_PROCESSING]);
    printf("MP Dump: %llu\n", h_times[MP_DUMP]);
    printf("Queue Write: %llu\n", h_times[QUEUE_WRITE]);
    */
    // printf("%llu\t%llu\t%llu\t%llu\t%llu\t\n", h_times[QUEUE_READ], h_times[MP_FETCH], h_times[MP_PROCESSING], h_times[MP_DUMP], h_times[QUEUE_WRITE]);

    hipFree(gq);
    // hipFree(d_change);
}


// Após a execução desse kernel, todo elemento do marker é menor ou igual que o elemento correspondente do mask
void __global__ clipImage(unsigned char* marker, unsigned char* mask, size_t pitchMarker, size_t pitchMask) {
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    int myCol = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned char* markerRow =  (unsigned char*)((char*)marker + (myRow) * pitchMarker);
    unsigned char* maskRow =  (unsigned char*)((char*)mask + (myRow) * pitchMask);

    markerRow[myCol] = min(markerRow[myCol], maskRow[myCol]);

}
